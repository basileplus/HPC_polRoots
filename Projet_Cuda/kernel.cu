#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <emmintrin.h>
#include <immintrin.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <intrin.h>
#include <omp.h>

#define ALIGNMENT	16			// Alignement des données en mémoire
#define	NB_POLYS	20000000	// Nombre de polynomes à résoudre

// Macro pour tester les codes d'erreur des fonctions Cuda
#define cudaCheckError(code,mess)	if (code != hipSuccess) printf("Cuda erreur (%s): %s\n", mess, hipGetErrorString(code))

typedef struct {
	float reel;
	float imaginaire;
} complexe_t;	// Type pour les complexes

// Les trois tableaux suivants contiennent les coefficients des polynômes.
float __declspec(align(ALIGNMENT)) coefs_A[NB_POLYS], coefs_B[NB_POLYS], coefs_C[NB_POLYS];
// Les deux tableaux suivants contiennent les solutions des polynômes précédents
complexe_t __declspec(align(ALIGNMENT)) Solutions_1[NB_POLYS], Solutions_2[NB_POLYS];

unsigned int maxThreadsPerBlock = 1;	// Nombre maximal de threads par bloc pour le GPU
unsigned int countMultiProcessor = 1;	// Nombre de multi-processeurs du GPU

// La fonction suivante affiche et met à jour certaines propriétés du GPU
void print_cuda_properties(void)
{
	int nb_devices;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&nb_devices);
	if (nb_devices == 0) return;

	hipGetDeviceProperties(&prop, 0);
	printf("%s - Version %2d.%2d\n", prop.name, prop.major, prop.minor);
	maxThreadsPerBlock = prop.maxThreadsPerBlock;
	countMultiProcessor = prop.multiProcessorCount;
}

// La fonction suivante initialise les tableaux coefs_A, coefs_B et coefs_C
void init_poly_coefs(void)
{
	int i;
	for (i = 0; i < NB_POLYS; i++)
	{
		do { coefs_A[i] = ((float)rand() / RAND_MAX) * 20.0 - 10.0; } while (coefs_A[i] == 0.0);
		coefs_B[i] = ((float)rand() / RAND_MAX) * 20.0 - 10.0;
		coefs_C[i] = ((float)rand() / RAND_MAX) * 20.0 - 10.0;
	}
}

// La fonction suivante met à zéro les solutions
void raz_solutions(void)
{
	int i;
	for (i = 0; i < NB_POLYS; i++)
	{
		Solutions_1[i].reel = Solutions_1[i].imaginaire = 0.0;
		Solutions_2[i].reel = Solutions_2[i].imaginaire = 0.0;
	}
}

// La fonction suivante affiche les racines des quatre derniers polynomes 
void print_results(char* mess)
{
	int i;
	for (i = NB_POLYS - 4; i < NB_POLYS; i++)
	{
		printf("%s\tPolynome n %d:\n", mess, i);
		printf("%s\t\t%2.4f . x^2 %c %2.4f . x %c %2.4f = 0.0\n", mess, coefs_A[i], (coefs_B[i] >= 0.0) ? '+' : '-', fabs(coefs_B[i]), (coefs_C[i] >= 0.0) ? '+' : '-', fabs(coefs_C[i]));
		printf("%s\t\t\tSolution 1 : %2.4f %c i . %2.4f\n", mess, Solutions_1[i].reel, (Solutions_1[i].imaginaire >= 0.0) ? '+' : '-', fabs(Solutions_1[i].imaginaire));
		printf("%s\t\t\tSolution 2 : %2.4f %c i . %2.4f\n", mess, Solutions_2[i].reel, (Solutions_2[i].imaginaire >= 0.0) ? '+' : '-', fabs(Solutions_2[i].imaginaire));
	}
}

// A compléter
void poly2_scalaire(float* A, float* B, float* C, complexe_t* Sols_1, complexe_t* Sols_2)
{
}

// A compléter
void poly2_scalaire_omp(float* A, float* B, float* C, complexe_t* Sols_1, complexe_t* Sols_2)
{
}

// A compléter
void poly2_sse2(float* A, float* B, float* C, complexe_t* Sols_1, complexe_t* Sols_2)
{
}

// A compléter
void poly2_sse2_omp(float* A, float* B, float* C, complexe_t* Sols_1, complexe_t* Sols_2)
{
}

// A compléter
__global__ void poly2_cuda(float* A, float* B, float* C, complexe_t* Sols_1, complexe_t* Sols_2)
{
}

int main()
{
	unsigned long long Debut, Fin, dureeScalaire, dureeScalaireOMP, dureeSSE, dureeSSEOMP, dureeCuda;
	float* ptr_Cuda_coefs_A, * ptr_Cuda_coefs_B, * ptr_Cuda_coefs_C;
	complexe_t* ptr_Cuda_sols_1, * ptr_Cuda_sols_2;
	hipError_t hipError_t;

	print_cuda_properties();
	init_poly_coefs();

	Debut = __rdtsc();
	poly2_scalaire(coefs_A, coefs_B, coefs_C, Solutions_1, Solutions_2);
	Fin = __rdtsc();
	dureeScalaire = Fin - Debut;
	print_results("Scal    ");

	Debut = __rdtsc();
	poly2_scalaire_omp(coefs_A, coefs_B, coefs_C, Solutions_1, Solutions_2);
	Fin = __rdtsc();
	dureeScalaireOMP = Fin - Debut;
	print_results("Scal OMP");

	Debut = __rdtsc();
	poly2_sse2(coefs_A, coefs_B, coefs_C, Solutions_1, Solutions_2);
	Fin = __rdtsc();
	dureeSSE = Fin - Debut;
	print_results("SSE2    ");

	Debut = __rdtsc();
	poly2_sse2_omp(coefs_A, coefs_B, coefs_C, Solutions_1, Solutions_2);
	Fin = __rdtsc();
	dureeSSEOMP = Fin - Debut;
	print_results("SSE2 OMP");

	cudaCheckError(hipMalloc(&ptr_Cuda_coefs_A, NB_POLYS * sizeof(float)), "hipMalloc - coefs_A");
	cudaCheckError(hipMalloc(&ptr_Cuda_coefs_B, NB_POLYS * sizeof(float)), "hipMalloc - coefs_B");
	cudaCheckError(hipMalloc(&ptr_Cuda_coefs_C, NB_POLYS * sizeof(float)), "hipMalloc - coefs_C");
	cudaCheckError(hipMalloc(&ptr_Cuda_sols_1, NB_POLYS * sizeof(complexe_t)), "hipMalloc - sols_1");
	cudaCheckError(hipMalloc(&ptr_Cuda_sols_2, NB_POLYS * sizeof(complexe_t)), "hipMalloc - sols_2");
	cudaCheckError(hipMemcpy(ptr_Cuda_coefs_A, coefs_A, NB_POLYS * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy - coefs_A");
	cudaCheckError(hipMemcpy(ptr_Cuda_coefs_B, coefs_B, NB_POLYS * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy - coefs_B");
	cudaCheckError(hipMemcpy(ptr_Cuda_coefs_C, coefs_C, NB_POLYS * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy - coefs_C");
	Debut = __rdtsc();
	poly2_cuda << <1, 1 >> > (ptr_Cuda_coefs_A, ptr_Cuda_coefs_B, ptr_Cuda_coefs_C, ptr_Cuda_sols_1, ptr_Cuda_sols_2);
	cudaCheckError(hipGetLastError(), "cuda Kernel poly2");
	hipDeviceSynchronize();
	Fin = __rdtsc();
	dureeCuda = Fin - Debut;

	cudaCheckError(hipMemcpy(Solutions_1, ptr_Cuda_sols_1, NB_POLYS * sizeof(complexe_t), hipMemcpyDeviceToHost), "hipMemcpy - sols_1");
	cudaCheckError(hipMemcpy(Solutions_2, ptr_Cuda_sols_2, NB_POLYS * sizeof(complexe_t), hipMemcpyDeviceToHost), "hipMemcpy - sols_2");
	cudaCheckError(hipFree(ptr_Cuda_coefs_A), "hipFree - coefs_A");
	cudaCheckError(hipFree(ptr_Cuda_coefs_B), "hipFree - coefs_B");
	cudaCheckError(hipFree(ptr_Cuda_coefs_C), "hipFree - coefs_C");
	cudaCheckError(hipFree(ptr_Cuda_sols_1), "hipFree - sols_1");
	cudaCheckError(hipFree(ptr_Cuda_sols_2), "hipFree - sols_2");

	print_results("Cuda");

	printf("Duree scalaire       : %lld cycles\n", dureeScalaire);
	printf("Duree scalaire OMP   : %lld cycles - Gain = %2.2f\n", dureeScalaireOMP, ((double)dureeScalaire) / dureeScalaireOMP);
	printf("Duree SSE            : %lld cycles - Gain = %2.2f\n", dureeSSE, ((double)dureeScalaire) / dureeSSE);
	printf("Duree SSE OMP        : %lld cycles - Gain = %2.2f\n", dureeSSEOMP, ((double)dureeScalaire) / dureeSSEOMP);
	printf("Duree cuda           : %lld cycles - Gain = %2.2f\n", dureeCuda, ((double)dureeScalaire) / dureeCuda);

	return 0;
}
